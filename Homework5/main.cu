#include <cstdio>
#include <iostream>
#include <string>
#include <vector>
#include <time.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>

#define BLOCK_SIZE 12288 //number of floats processed per SM

#define MAX_THREADS_PER_BLOCK 1024 //Limit of GTX 1080
#define BLOCK_HEIGHT 32
#define BLOCK_WIDTH 32 
#define ELM_PER_THREAD 12 // BLOCK_SIZE / MAX_THREADS

using namespace std;

timespec start_time;
timespec stop_time;
void start_clock();
void stop_clock();
double get_clock_result_seconds();
void print_time_seconds(double seconds);

void PrintPartialMatrix(size_t n, float* matrix)
{
    if(n < 5)
    {
        printf("Matrix is too small to print.\n");
        return;
    }

    for(size_t i = 0; i < 5; ++i)
    {
        for(size_t j = 0; j < 5; j++)
        {
            printf("%0.2f\t", matrix[(i*n) + j]);
        }
        printf("\n");
    }
}

__global__ void transpose_one_to_one(size_t n, float* input, float* output)
{
    int global_col = (blockDim.x * blockIdx.x) + threadIdx.x;
    int global_row = (blockDim.y * blockIdx.y) + threadIdx.y;
    int i = (n * global_row) + global_col;
    //printf("block(%i, %i)\tthread(%i, %i)\ti:%i\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, i);
    if(i >= n*n)
        return;
    
    output[((i % n)*n) + (i / n)] = input[i];
}
__global__ void transpose_optimized(size_t n, float* input, float* output)
{
    __shared__ float s_data[BLOCK_SIZE];
    
    //unsigned int global_col = ((blockDim.x * blockIdx.x) * ELM_PER_THREAD) + (threadIdx.x * ELM_PER_THREAD);
    //unsigned int global_row = (blockDim.y * blockIdx.y) + threadIdx.y;
    //unsigned int i = (n * (global_row)) + (global_col);
    unsigned int i = (n * ((blockDim.y * blockIdx.y) + threadIdx.y)) + (((blockDim.x * blockIdx.x) * ELM_PER_THREAD) + (threadIdx.x * ELM_PER_THREAD));

    unsigned int block_level_index = ((threadIdx.y * blockDim.x) + threadIdx.x) * ELM_PER_THREAD;

    unsigned int start = i + block_level_index;
    unsigned int stop = start + ELM_PER_THREAD;

    int s_idx = block_level_index;
    for(unsigned int i = start; i < stop; i++, s_idx++)
    {   
        if(i >= n*n)
            break;
        s_data[s_idx] = input[i];
    }
    __syncthreads();

    s_idx = block_level_index;
    for(int i = start; i < stop; i++, s_idx++)
    {
        if(i >= n*n)
            break;
        output[((i % n)*n) + (i / n)] = s_data[s_idx];
    }
}

int main(int argc, char** argv)
{
    if (argc < 2)
    {
        printf("Not enough arguments\n");
        printf("Usage is ./a.out [matrix dim]\n");
        return 1;
    }

    string dimension_arg = argv[1];
    size_t N = 0;
    try{
        N = strtoul(dimension_arg.c_str(), NULL, 10);
    }
    catch(...){
        printf("Matrix dimension argument %s is not valid\n", dimension_arg.c_str());
        return 2;
    }

    size_t matrix_size = N*N*sizeof(float);

    float* d_input_matrix;
    float* d_resultant_matrix;
    hipMalloc((void **)&d_input_matrix, matrix_size);
    hipMalloc((void **)&d_resultant_matrix, matrix_size);

    float* h_input_matrix = new float[N*N];
    float* h_resultant_matrix_1 = new float[N*N];
    float* h_resultant_matrix_2 = new float[N*N];
    srand(time(NULL));
    for( int i = 0; i < N*N; i++)
        h_input_matrix[i] = (float)rand() / (float)RAND_MAX;
        //h_input_matrix[i] = 0.11;

    hipMemcpy(d_input_matrix, h_input_matrix, matrix_size, hipMemcpyHostToDevice);


    dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);

    size_t grid_width = N / BLOCK_WIDTH;
    grid_width += N % BLOCK_WIDTH > 0 ? 1 : 0;

    size_t grid_height = N / (BLOCK_HEIGHT);
    grid_height += N % BLOCK_HEIGHT > 0 ? 1 : 0;

    dim3 grid(grid_width, grid_height); 

    printf("grid(%lu, %lu)\n", grid_width, grid_height);

    start_clock();
    transpose_one_to_one<<<grid, block>>>(N, d_input_matrix, d_resultant_matrix);
    hipDeviceSynchronize();
    stop_clock();
    printf("naive time:\t");
    print_time_seconds(get_clock_result_seconds());
    printf("\n\n");
    hipMemcpy(h_resultant_matrix_1, d_resultant_matrix, matrix_size, hipMemcpyDeviceToHost);




    block = dim3(BLOCK_WIDTH, BLOCK_HEIGHT);

    grid_width = N / (BLOCK_WIDTH * ELM_PER_THREAD);
    grid_width += N % BLOCK_WIDTH > 0 ? 1 : 0;

    grid_height = N / (BLOCK_HEIGHT);
    grid_height += N % BLOCK_HEIGHT > 0 ? 1 : 0;

    grid = dim3(grid_width, grid_height); 

    printf("grid(%lu, %lu)\n", grid_width, grid_height);

    start_clock();
    transpose_optimized<<<grid, block>>>(N, d_input_matrix, d_resultant_matrix);
    hipDeviceSynchronize();
    stop_clock();
    printf("optimized time:\t");
    print_time_seconds(get_clock_result_seconds());
    printf("\n");

    hipMemcpy(h_resultant_matrix_2, d_resultant_matrix, matrix_size, hipMemcpyDeviceToHost);


    if (memcmp(h_resultant_matrix_1, h_resultant_matrix_2, matrix_size) != 0)
    {
        printf("Results DO NOT match!\n");\
        for(size_t i = 0; i < matrix_size/sizeof(float); i++)
        {
            if(h_resultant_matrix_1[i] != h_resultant_matrix_2[i])
            {
                printf("index %lu doesn't match\n", i);
                printf("--1: %0.2f\t2: %0.2f--\n", h_resultant_matrix_1[i], h_resultant_matrix_2[i]);
                printf("Input:\n");
                PrintPartialMatrix(N, h_input_matrix);
                printf("\nOutput 1:\n");
                PrintPartialMatrix(N, h_resultant_matrix_1);
                printf("\nOutput 2:\n");
                PrintPartialMatrix(N, h_resultant_matrix_2);
                break;
            }
        }
    }
    else 
    {
        printf("Results match\n");
    }

    hipFree(d_input_matrix);
    hipFree(d_resultant_matrix);
    delete[] h_input_matrix;
    delete[] h_resultant_matrix_1;
    delete[] h_resultant_matrix_2;


    return 0;
}

void start_clock()
{
	clock_gettime(CLOCK_THREAD_CPUTIME_ID, &start_time);
}
void stop_clock()
{
	clock_gettime(CLOCK_THREAD_CPUTIME_ID, &stop_time);
}
double get_clock_result_seconds()
{
	double result = stop_time.tv_sec - start_time.tv_sec;
	result += (double)(stop_time.tv_nsec - start_time.tv_nsec) / 1000000000;
	return result;
}
void print_time_seconds(double seconds)
{
#ifdef _WIN32
	printf("%0.3f seconds", seconds);
#elif __linux__
	printf("%0.9f seconds", seconds);
#endif
}
